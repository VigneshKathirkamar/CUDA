#include "hip/hip_runtime.h"
#include<stdio.h>

__global__ void mykernel()
{
    printf("Hello World, I started CUDA\n");
}

int main()
{
 mykernel<<<1,10>>>();
 hipDeviceSynchronize();
 //printf("Hello world \n");
 return 0;
}

//This will print "Hello World, I started CUDA" 10 times since we have instantiated 10 threads on Kernel launch parameter <1,10>
" 10 times since we have instantiated 10 threads on Kernel launch parameter <1,10>
